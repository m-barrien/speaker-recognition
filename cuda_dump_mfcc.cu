#include "hip/hip_runtime.h"
/*
FUNCION PARA CONVERTIR UN BUFFER DE SEÑAL A COEFICIENTES DE ESPECTRO MEL MFCC

*/

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <time.h>
#include <signal.h>
#include "CUDAPreprocessing.h"
#include <thread>
#include <string>
#include <mutex>
#include <iostream>
#include <cstdint>
#include "conf.h"

static float *float_buffer;

//seconds * SAMPLES_PER_SECOND/RAW_PERIOD_SAMPLE_SIZE
static int frames_to_process = 10;
static int frames_to_process_i = frames_to_process;

void exit_handler(int sig){
  if (sig == SIGINT)
  {
    /* code */
  }
  frames_to_process=0;
}

int main() {
  //Señal para terminar
  signal (SIGINT,exit_handler);

  float_buffer = new float[RAW_PERIOD_SAMPLE_SIZE*N_CHANNELS];

  CUDASignalPreprocessor sProcessor =  CUDASignalPreprocessor(float_buffer, RAW_PERIOD_SAMPLE_SIZE, FRAME_SIZE, WINDOW_OVERLAP, SAMPLES_PER_SECOND);

  sProcessor.buildFilterBanks(N_FILTERS,MIN_FREQ,MAX_FREQ); 
  sProcessor.configureMFCC(N_MFCC_COEFS); //n mfcc coefs

  //output mfcc buffer
  int n_mfcc_frames =sProcessor.getFrameCount();
  int n_mfcc_coefs =sProcessor.getMfccCount();
  float *mfcc_buffer = new float[n_mfcc_frames * n_mfcc_coefs];

  int time_s = clock();
  while(frames_to_process >= 0){

    for (int i = 0; i < RAW_PERIOD_SAMPLE_SIZE*N_CHANNELS; ++i)
    {
      float_buffer[i] = ((float)rand()/(float)(RAND_MAX)) * 2 - 1;; //randomsignal
    }
    //convert signal to mfcc coefs and write to mfcc_buffer
    sProcessor.getMfccCoefs(&n_mfcc_frames,&n_mfcc_coefs,mfcc_buffer);
    
    //write(1, mfcc_buffer, sizeof(float)*n_mfcc_frames*n_mfcc_coefs);
    frames_to_process--;
  }
  double runtime = (double)(clock() - time_s) / (double)CLOCKS_PER_SEC;
  std::cout << runtime << " segundos para procesar " << frames_to_process_i << " frames de audio o " << frames_to_process_i * RAW_PERIOD_SAMPLE_SIZE/SAMPLES_PER_SECOND << " segundos de audio real" <<std::endl;
  return 0;
}

