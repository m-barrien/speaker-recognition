#include "hip/hip_runtime.h"
#include "CUDAPreprocessing.h"

__constant__ wave_params_t dev_wave_params;

__device__ __forceinline__ float  hamming_window(int j){
	return ( 0.54 - 0.46 * cos((2*M_PI*j)/(dev_wave_params.frame_len-1)) );
}
__device__ void dev_applyPreEmphasis(float emph){

}
__device__ void dev_dumpToFrames(){

}
__device__ void dev_applyWindowsToFrames(){

}
__device__ void dev_framesFFTtoPowSpec(){

}
__device__ void dev_powerFramesToLogEnergies(){

}
__device__ void dev_logEnergyToMFCC(){

}
__global__ void kernel_getMfccCoefs(){
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < 1){
		printf("%d %d\n", tId, dev_wave_params.sample_rate);
	}
}

CUDASignalPreprocessor::CUDASignalPreprocessor(float* buffer,int buffer_len, int _frame_len, float overlap_percentage, int sr){
	assert(overlap_percentage >= 0.f && overlap_percentage < 1.f);
	assert(buffer_len >= _frame_len);
	this->wave_params.signal_buffer = buffer;
	hipMalloc(&(this->wave_params.dev_signal_buffer), sizeof(float)*buffer_len);
	this->wave_params.raw_buffer_len = buffer_len;
	this->wave_params.frame_len = _frame_len;
	this->wave_params.in_frame_offset = (int) (_frame_len*(1.0f - overlap_percentage));

	int i=0;
	while(buffer_len >= _frame_len + i*this->wave_params.in_frame_offset) i++;
	this->wave_params.frame_count = i;

	hipMalloc(&(this->wave_params.frames), sizeof(float)*this->wave_params.frame_count*_frame_len);
	hipMalloc(&(this->wave_params.power_frames), sizeof(float)*this->wave_params.frame_count*_frame_len);

	/*
		KISS FFT allocate
		*/
	//this->wave_params.fft_cfg = kiss_fftr_alloc(_frame_len,0,NULL, NULL);
	this->wave_params.sample_rate = sr;
	this->wave_params.base_freq = (float)sr/(float)this->wave_params.frame_len;

}
void CUDASignalPreprocessor::getMfccCoefs(int* nframes,int* pN_mfcc_coefficients,float* output){
	assert(nframes && pN_mfcc_coefficients && output != NULL);
	hipMemcpy(this->wave_params.dev_signal_buffer, this->wave_params.signal_buffer, sizeof(float)*this->wave_params.raw_buffer_len, hipMemcpyHostToDevice);
	kernel_getMfccCoefs<<<1,1024>>>();
	hipDeviceSynchronize();
	/*
	this->applyPreEmphasis(0.97f);
    this->dumpToFrames();
    this->applyWindowsToFrames();
    this->framesFFTtoPowSpec();
    this->powerFramesToLogEnergies();
    this->logEnergyToMFCC();

    *nframes = this->wave_params.frame_count;
    *pN_mfcc_coefficients = this->wave_params.n_mfcc_coefficients;
    memcpy(output, this->wave_params.mfcc_frames, sizeof(float)*this->wave_params.frame_count*this->wave_params.n_mfcc_coefficients);
    */
}

int CUDASignalPreprocessor::getFrameCount(void){
	return this->wave_params.frame_count;
}
int CUDASignalPreprocessor::getMfccCount(void){
	return this->wave_params.n_mfcc_coefficients;
}

void CUDASignalPreprocessor::applyPreEmphasis(float preemphasis){
	for ( 
		int i = this->wave_params.raw_buffer_len-1 ; 
		i > 0; 
		i--
		)
	{
		this->wave_params.signal_buffer[i] = this->wave_params.signal_buffer[i] - preemphasis* this->wave_params.signal_buffer[i-1];
	}
}
void CUDASignalPreprocessor::dumpToFrames(void){
	for (int i = 0; i < this->wave_params.frame_count; ++i)
	{
		memcpy(this->wave_params.frames[i], this->wave_params.signal_buffer + i*this->wave_params.in_frame_offset, this->wave_params.frame_len*sizeof(float)); //4 bytes per float
	}
}
void CUDASignalPreprocessor::applyWindowsToFrames(void){
	float window_buffer[this->wave_params.frame_len];

	for (int i = 0; i < this->wave_params.frame_count; ++i)
	{
		for (int j = 0; j < this->wave_params.frame_len; ++j)
		{
			window_buffer[j]=this->wave_params.frames[i][j] * this->wave_params.hamming_window[j];
		}
		memcpy(this->wave_params.frames[i], window_buffer, this->wave_params.frame_len*sizeof(float)); 
	}
}
void CUDASignalPreprocessor::framesFFTtoPowSpec(void){
//	kiss_fft_cpx complex_arr[this->wave_params.frame_len];
	for (int i = 0; i < this->wave_params.frame_count; ++i)
	{
		//kiss_fftr(this->wave_params.fft_cfg, this->wave_params.frames[i], complex_arr);
		for (int j = 0; j < this->wave_params.frame_len; ++j)
		{
			//this->wave_params.power_frames[i][j] = (pow(complex_arr[j].r,2) + pow(complex_arr[j].i,2))/this->wave_params.frame_len;
		}
	}
}
inline float CUDASignalPreprocessor::melToHz(float mel){
	return 700*(exp(mel/1125)-1);
}
inline float CUDASignalPreprocessor::hzToMel(float f){
	return 1125*log(1+f/700);
}
void CUDASignalPreprocessor::buildFilterBanks(int nfilters,int f0, int fmax){
	this->wave_params.mel_values = new float[nfilters+2];
	this->wave_params.freq_values = new float[nfilters+2];
	this->wave_params.n_mel_filters = nfilters;

	this->wave_params.log_energy_frames = new float*[this->wave_params.frame_count];
	for (int j = 0; j < this->wave_params.frame_count; ++j)
	{
		this->wave_params.log_energy_frames[j] = new float[this->wave_params.n_mel_filters];
	}

	float mel_i = hzToMel(f0);
	float mel_f = hzToMel(fmax);
	for (int i = 0; i < nfilters+2; ++i)
	{
		this->wave_params.mel_values[i] = mel_i + i*(mel_f - mel_i)/(nfilters+1);
		this->wave_params.freq_values[i] = melToHz(this->wave_params.mel_values[i]);
	}
}
float CUDASignalPreprocessor::filterValue(int bank_index, float power_freq){
	assert(this->wave_params.freq_values != NULL && bank_index <= this->wave_params.n_mel_filters && bank_index >0);
	if (power_freq < this->wave_params.freq_values[bank_index-1]) return 0.f;
	else if (
		power_freq > this->wave_params.freq_values[bank_index-1]
		&&
		power_freq <= this->wave_params.freq_values[bank_index]
	)
	{
		return (power_freq -this->wave_params.freq_values[bank_index-1])/(this->wave_params.freq_values[bank_index]-this->wave_params.freq_values[bank_index-1]);
	}
	else if (
		power_freq > this->wave_params.freq_values[bank_index]
		&&
		power_freq <= this->wave_params.freq_values[bank_index+1]
		)
	{
		return ( this->wave_params.freq_values[bank_index+1] - power_freq )/(this->wave_params.freq_values[bank_index+1]-this->wave_params.freq_values[bank_index]);
	}
	else return 0.f;

}

void CUDASignalPreprocessor::powerFramesToLogEnergies(void){
	for (int i = 0; i < this->wave_params.frame_count; ++i)
	{
		for (int filter_index = 0; filter_index < this->wave_params.n_mel_filters; ++filter_index)
		{
			this->wave_params.log_energy_frames[i][filter_index]=0;
			for (int j = 0;
				j < 1 + this->wave_params.frame_len/2 
				&& 
				j*this->wave_params.base_freq< this->wave_params.freq_values[this->wave_params.n_mel_filters+1]; //gone past filter frequency so all ceroes
				++j)
			{
				float actual_freq = j * this->wave_params.base_freq;
				this->wave_params.log_energy_frames[i][filter_index] += this->wave_params.power_frames[i][j] * filterValue(filter_index+1,actual_freq);
			}
			this->wave_params.log_energy_frames[i][filter_index] = 20 * log10( this->wave_params.log_energy_frames[i][filter_index] );
		}
	}
}

void CUDASignalPreprocessor::configureMFCC(int _n_mfcc_coefficients){
	assert(_n_mfcc_coefficients <= this->wave_params.n_mel_filters);
	this->wave_params.n_mfcc_coefficients = _n_mfcc_coefficients;
	this->wave_params.mfcc_frames = new float[this->wave_params.frame_count * _n_mfcc_coefficients];

	hipMemcpyToSymbol(HIP_SYMBOL(&dev_wave_params), &(this->wave_params), sizeof(wave_params_t), 0, hipMemcpyHostToDevice);
}
void CUDASignalPreprocessor::logEnergyToMFCC(void){
	float C_n =0;
	for (int f_indx = 0; f_indx < this->wave_params.frame_count; ++f_indx)
	{
		for (int n = 0; n < this->wave_params.n_mfcc_coefficients; ++n)
		{
			C_n =0;
			for (int k = 0; k < this->wave_params.n_mfcc_coefficients; ++k)
			{
				C_n += this->wave_params.log_energy_frames[f_indx][k] * cos(n*(k-0.5f)*M_PI/this->wave_params.n_mfcc_coefficients);
			}
			this->wave_params.mfcc_frames[this->wave_params.n_mfcc_coefficients*f_indx + n] = C_n;
		}
	}
}

